// COMPILE: nvcc thrustSortSimple.cu -I cuda_common/inc/ -o thrustSort

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cuda_common.h>     

static void cuda_assert(const hipError_t code, const char* const file, const int line, const bool abort) {
    if (code != hipSuccess) {
        fprintf(stderr,"cuda_assert: %s %s %d\n",hipGetErrorString(code),file,line);

        if (abort) {
            hipDeviceReset();          
            exit(code);
        }
    }
}

#define cuda( ...) { cuda_assert((cuda##__VA_ARGS__), __FILE__, __LINE__, true ); }

static void sort(thrust::host_vector<uint64_t>& h_vec, hipEvent_t start, hipEvent_t end, float* const elapsed) {
    
    cuda(  EventRecord (start, nullptr)  );

    thrust::device_vector<uint64_t> d_vec = h_vec; // copy data to device
    thrust::sort(d_vec.begin(), d_vec.end()); // sort data on device 

    cuda( EventRecord(end, nullptr) );
    cuda( EventSynchronize(end) );

    float sort_elapsed;
    cuda(  EventElapsedTime(&sort_elapsed, start, end)  );
    *elapsed += sort_elapsed;
}

static void measure(const struct hipDeviceProp_t* const props, const uint32_t DATASIZE) {
    
    thrust::host_vector<uint64_t> h_vec(DATASIZE);
    std::generate(h_vec.begin(), h_vec.end(), rand);

    hipEvent_t start, end;
    cuda( EventCreate(&start) );
    cuda( EventCreate(&end) );

    float elapsed = 0.0f;
    for (int a=0; a<20; a++) sort(h_vec, start, end, &elapsed);

    cuda( EventDestroy(start) );
    cuda( EventDestroy(end) );

    float time = elapsed / 20.0 / 1000; // in secs
    printf("Throughput =%9.3lf MElements/s, Time = %.3lf ms\n", 1e-6*DATASIZE/time, time*1000);
}

int main(int argc, char** argv) {

    DisplayCudaDevice();
    
    struct hipDeviceProp_t props;
    const uint32_t DATASIZE = (8<<20); // 32M
    measure( &props, DATASIZE );  // SORT
    cuda( DeviceReset() ); // RESET

    return 0;
}
