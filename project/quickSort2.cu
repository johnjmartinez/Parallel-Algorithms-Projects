#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

void printArray(int *a, int len, const char *fileName) {
    FILE *fptr;
    fptr = fopen(fileName, "w");
    for (int i = 0; i < len; i++) fprintf(fptr, "%d\n", a[i]);
    fclose(fptr);
}

__device__ int d_size;

//swap function
__device__ void swap(int *x,int *y) { 
    int temp = *x;
    *x = *y;
    *y = temp;
}

__global__ void partition (int *arr, int *arr_l, int *arr_h, int n) {
    
    int myId = blockIdx.x * blockDim.x + threadIdx.x;
    d_size = 0;

    __syncthreads();

    if(myId >=n) return;

    if (myId < n) {
        int end = arr_h[myId];
        int begin = arr_l[myId];

        int pivot = arr[end];
        int p_index = (begin - 1);

        for (int i = begin; i <= end- 1; i++) {
            if (arr[i] <= pivot) {
                p_index++;
                swap( &arr[i], &arr[p_index] );
            }
        }

        swap( &arr[p_index+1], &arr[end]);

        int j = (p_index + 1);
        if (j-1 > begin) {
            int index = atomicAdd(&d_size, 1);
            arr_l[index] = begin;
            arr_h[index] = j-1;
        }
        if ( j+1 < end ) {
            int index = atomicAdd(&d_size, 1);
            arr_l[index] = j+1;
            arr_h[index] = end;
        }
    }
}

//quicksort
float quickSort (int h_input[], int begin, int end) {

    int nthreads;
    int nblocks;

    float et=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int h_low[end - begin + 1], h_high[end - begin + 1];  //pointers

    int top = -1, *d_input, *d_low, *d_high;

    h_low[ ++top ] = begin;
    h_high[ top ] = end;

    int arr_size = (end+1)*sizeof(int);
    int mysize;
    int k=1;

    hipMalloc((void **) &d_input, arr_size);
    hipMemcpy(d_input, h_input,arr_size, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_low, arr_size);
    hipMemcpy(d_low, h_low, arr_size, hipMemcpyHostToDevice);

    hipMalloc((void **) &d_high, arr_size);
    hipMemcpy(d_high, h_high, arr_size, hipMemcpyHostToDevice);

    nthreads = 1;
    nblocks = 1;

    hipEventRecord(start);
    while ( k > 0 ) {
        partition<<< nblocks, nthreads >>>( d_input, d_low, d_high, k );
        hipMemcpyFromSymbol(&mysize, HIP_SYMBOL(d_size), sizeof(int), 0, hipMemcpyDeviceToHost);

        if (mysize < 1024) 
            nthreads = mysize;
        else {
            nthreads = 1024;
            nblocks = mysize/nthreads + (mysize%nthreads == 0 ? 0 : 1);
        }
        
        k = mysize;
        //cudaEventRecord(stop);
        //cudaEventSynchronize(stop);    
    }
    
    hipMemcpy(h_input, d_input,arr_size,hipMemcpyDeviceToHost);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&et, start, stop);
    return et;
}

int main(int argc, char **argv) {

    int DATASIZE = atoi(argv[1]); 
    int h_input[DATASIZE];
    srand(time(NULL));

    //allocating host array with random values
    for (int i = 0; i<DATASIZE; i++) h_input[i] = rand ();

    float et = quickSort(h_input, 0, DATASIZE-1);
    printf("%d Elements, Time : %.3lf ms\n", DATASIZE, et);

    
    //testing sort
    for(int i=0;i<DATASIZE-1;i++) {
        if (h_input[i]>h_input[i+1]) {
            printf("Sorting Failed\n");
            break;
        }
        else if(i== DATASIZE-2) printf("Sorting Successful\n");
    }
    
    //printing result printArray( h_input, DATASIZE, "output");
    return 0;
}
