#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define DATASIZE    1000000
#define BLOCK_SIZE  512

void printArray(int a[],int n) {
    FILE *fptr;
    fptr = fopen("Brick_sort_result.txt", "w");

    for(int i=0; i < n; i++) {
        fprintf(fptr,"%d ",a[i]);

	if((i+1)%10 == 0)
	    fprintf(fptr,"\n");
    }

    fclose(fptr);
}

__global__ void oddevensort ( int * input, unsigned int size, int i ) {

    unsigned int myId = threadIdx.x + blockDim.x * blockIdx.x;

    int temp;

    if(myId > size)
	return;

    if( i == 0 ) {
	//For even threads
	if(( myId % 2 == 0 && input[myId] > input[myId+1])) {
	    temp = input[myId+1];
	    input[myId+1] = input[myId];
	    input[myId] = temp;
	}
    }
    else {
	//for odd threads
	if(( myId % 2 != 0 && input[myId] > input[myId+1])) {
	    temp = input[myId+1];
	    input[myId+1] = input[myId];
	    input[myId] = temp;
	}
    }

    __syncthreads();
}


int main() {

    float et=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    unsigned int arr_size = DATASIZE * sizeof(int);
    
    //host arrays
    int h_input[DATASIZE], h_output[DATASIZE];
    
    //device arrays
    int *d_input, *d_output;
    
    //allocating memory for device arrays
    hipMalloc((void**)&d_input, arr_size );
    hipMalloc((void**)&d_output, arr_size );
    
    srand(time(NULL));
    
    //generating host array values
    for( int i = 0; i < DATASIZE; i++ )
	h_input[i] = rand()%999;
    
    printf("Input array size : %d:\n",DATASIZE);
    if(DATASIZE<=100) {
	for( int i = 0; i < DATASIZE; i++ )
	    printf("%d ", h_input[i] );
    }
    printf("\n");
    
    
    //copy from host to device
    hipMemcpy( d_input, h_input, arr_size, hipMemcpyHostToDevice);
    
    int nthreads( BLOCK_SIZE );
    int nblocks( ceil((DATASIZE-1)/(float)BLOCK_SIZE) + 1 );
    
    hipEventRecord(start);
    
    for( int i=0; i<DATASIZE; i++)
	oddevensort<<< nblocks,nthreads >>>( d_input, DATASIZE, i%2 );
    
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    hipEventElapsedTime(&et, start, stop);
    printf("Time is: %f\n",et);
    
    hipMemcpy( h_output, d_input, arr_size, hipMemcpyDeviceToHost);
    
    printArray(h_output,DATASIZE);
    
    printf("\n");
    
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
