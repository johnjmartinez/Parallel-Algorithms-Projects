#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <cuda_common.h>    
#include <hipcub/hipcub.hpp>

// Parameters
double MIN_BENCH_TIME = 1.5;  // mimimum seconds to run each bechmark

template <typename T>
__global__ void randFill (T *d_array, uint32_t size) {

    const uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x);
    if (idx >= size)  return;
    
    // https://en.wikipedia.org/wiki/Linear_congruential_generator
    uint32_t rnd = idx*1234567891u;

    rnd = 29943829*rnd + 1013904223;    
    rnd = 29943829*rnd + 1013904223;

    uint64_t rnd1 = rnd;

    rnd = 29943829*rnd + 1013904223;
    rnd = 29943829*rnd + 1013904223;

    d_array[idx] = T(rnd1<<32) + rnd;
}

template <typename elem>
double devRadixSort (int SORT_BYTES, size_t n, void *d_array0, hipEvent_t &start, hipEvent_t &stop) {
    
    int begin_bit = 0,  end_bit = SORT_BYTES*8; // Bit subrange [begin_bit, end_bit) of differentiating elem bits
    auto d_array = (elem*) d_array0;

    hipcub::DoubleBuffer<elem> d_elems (d_array, d_array + n); // Create DoubleBuffer to wrap pair of Dev pointers

    // Determine temp Dev storage requirements
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    checkCudaErrors(hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_elems, n, begin_bit, end_bit));
    checkCudaErrors(hipMalloc (&d_temp_storage, temp_storage_bytes)); // Allocate temp storage

    int numIterations = 0;
    double totalTime = 0;
    
    checkCudaErrors(hipDeviceSynchronize());
    for ( ; totalTime < MIN_BENCH_TIME; numIterations++) {

        checkCudaErrors(hipEventRecord (start));
        randFill<elem> <<< n/1024+1, 1024 >>> (d_array, n);  // Fill source buffer with random numbers
        // Run sorting operation
        checkCudaErrors(hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_elems, n, begin_bit, end_bit));        
        checkCudaErrors(hipEventRecord (stop));
        checkCudaErrors(hipEventSynchronize(stop));
       
        float time;
        checkCudaErrors(hipEventElapsedTime (&time, start, stop));
        totalTime += time/1000; 
    }
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipFree (d_temp_storage)); // Release temp storage
    return totalTime/numIterations;
}

int main (int argc, char **argv) {

    const int DATASIZE = atoi(argv[1]); 

    DisplayCudaDevice();

    void* d_array;
    checkCudaErrors(hipMalloc(&d_array, 4*DATASIZE*sizeof(int)));

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    auto print = [&] (int bytes, int elemsize, double totalTime) {
        printf("%d/%d: Throughput =%9.3lf MElements/s, Time = %.3lf ms\n",
               bytes, elemsize, 1e-6 * DATASIZE / totalTime, totalTime * 1000);
    };

    printf("Sorting %d elements:\n", DATASIZE);
    for(int i=1; i<=4; i++)  
        print (i, 4, devRadixSort<int>(i, DATASIZE, d_array, start, stop));  
    hipDeviceReset();
    return 0;
}
