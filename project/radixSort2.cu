#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <thrust/host_vector.h>
#include <cuda_common.h>

#define BLOCK_SIZE 1024

#define CUDA_CHECK(val) cudaCheck( (val), #val, __FILE__, __LINE__)

template<typename T>
void cudaCheck(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

__global__ void checkBit(int* const d_inVals, int* const d_outPredct, const int bit, const size_t numElems) {
    // Predicate returns TRUE if significant bit is not present
    const int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= numElems) return;
    int Predct = ((d_inVals[id] & bit) == 0);
    d_outPredct[id] = Predct;
}

__global__ void flipBit(int* const d_list, const size_t numElems) { 
    const int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= numElems) return;
    d_list[id] = ((d_list[id] + 1) % 2);
}

__global__ void partnExclBlellochScan(int* const d_list, int* const d_blk_sums,  const size_t numElems) { 
    extern __shared__ int s_block_scan[];
    const int tid = threadIdx.x;
    const int id = blockDim.x * blockIdx.x + tid;

    // copy to shared memory, pad block if too small
    if (id >= numElems) s_block_scan[tid] = 0;
    else s_block_scan[tid] = d_list[id];
    
    __syncthreads();

    // reduce
    int i;
    for (i = 2; i <= blockDim.x; i <<= 1) {
      if ((tid + 1) % i == 0) {
        int neighbor_offset = i>>1;
        s_block_scan[tid] += s_block_scan[tid - neighbor_offset];
      }
      __syncthreads();
    }
    i >>= 1; // return i to last value before for-loop exit
    
    if (tid == (blockDim.x-1)) {
      d_blk_sums[blockIdx.x] = s_block_scan[tid];
      s_block_scan[tid] = 0; // set last (sum of whole block) to 0
    }
    
    __syncthreads();

    // downsweep
    for (i = i; i >= 2; i >>= 1) {
      if((tid + 1) % i == 0) {
        int neighbor_offset = i>>1;
        int old_neighbor = s_block_scan[tid - neighbor_offset];
        s_block_scan[tid - neighbor_offset] = s_block_scan[tid]; // copy
        s_block_scan[tid] += old_neighbor;
      }
      __syncthreads();
    }

    // copy result to global memory
    if (id < numElems) d_list[id] = s_block_scan[tid];
}

__global__ void scanAddBlkSums(int* const d_predctScan, int* const d_blkSumScan, const size_t numElems) { 
    const int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= numElems) return;
    d_predctScan[id] += d_blkSumScan[blockIdx.x];
}

__global__ void scatter(int* const d_in, int* const d_out, int* const d_predctTscan, int* const d_predctFscan,
                        int* const d_predctFalse, int* const d_numPredctTelems, const size_t numElems) { 

    const int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= numElems) return;

    int newLoc;
    if (d_predctFalse[id] == 1)  newLoc = d_predctFscan[id] + *d_numPredctTelems;
    else newLoc = d_predctTscan[id];

    d_out[newLoc] = d_in[id];
}

int* d_predct;
int* d_predctTscan;
int* d_predctFscan;
int* d_numPredctTelems; // GLOBAL POINTERS
int* d_numPredctFelems;
int* d_blk_sums;

void radixSort(int* const d_inVals, int* const d_inPos, int* const d_outVals, int* const d_outPos, 
               const size_t numElems) { 

  int blockSize = BLOCK_SIZE;
  size_t size = sizeof(int) * numElems;
  int gridSize = ceil(float(numElems) / float(blockSize));

  CUDA_CHECK(hipMalloc((void**)&d_predct, size));
  CUDA_CHECK(hipMalloc((void**)&d_predctTscan, size));
  CUDA_CHECK(hipMalloc((void**)&d_predctFscan, size));
  CUDA_CHECK(hipMalloc((void**)&d_numPredctTelems, sizeof(int)));
  CUDA_CHECK(hipMalloc((void**)&d_numPredctFelems, sizeof(int))); // throwaway
  CUDA_CHECK(hipMalloc((void**)&d_blk_sums, gridSize*sizeof(int)));

  size_t blkAlloc = sizeof(int)*blockSize;
  size_t gridAlloc = sizeof(int)*gridSize;

  int bitPosition;
  int max_bits = 31;
  for (int bit = 0; bit < max_bits; bit++) {
    bitPosition = 1<<bit;

    // create PredctTrue
    if ((bit + 1) % 2 == 1) 
      checkBit<<<gridSize, blockSize>>>(d_inVals, d_predct, bitPosition, numElems);
    else 
      checkBit<<<gridSize, blockSize>>>(d_outVals, d_predct, bitPosition, numElems);
    
    // scan Predct=True
    CUDA_CHECK(hipMemcpy(d_predctTscan, d_predct, size, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemset(d_blk_sums, 0, gridAlloc));

    partnExclBlellochScan<<<gridSize, blockSize, blkAlloc>>>(d_predctTscan, d_blk_sums, numElems);
    partnExclBlellochScan<<<1, blockSize, gridAlloc>>>(d_blk_sums, d_numPredctTelems, gridSize);
    scanAddBlkSums<<<gridSize, blockSize>>>(d_predctTscan, d_blk_sums, numElems);

    // transform Predct=True -> Predct=False
    flipBit<<<gridSize, blockSize>>>(d_predct, numElems);

    // scan Predct=False
    CUDA_CHECK(hipMemcpy(d_predctFscan, d_predct, size, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemset(d_blk_sums, 0, gridAlloc));

    partnExclBlellochScan<<<gridSize, blockSize, blkAlloc>>>(d_predctFscan, d_blk_sums, numElems);
    partnExclBlellochScan<<<1, blockSize, gridAlloc>>>(d_blk_sums, d_numPredctFelems, gridSize);
    scanAddBlkSums<<<gridSize, blockSize>>>(d_predctFscan, d_blk_sums, numElems);

    // scatter values (flip input/output depending on iteration)
    if ((bit + 1) % 2 == 1) {
      scatter<<<gridSize, blockSize>>>(d_inVals, d_outVals, d_predctTscan, d_predctFscan, d_predct, d_numPredctTelems, numElems);
      scatter<<<gridSize, blockSize>>>(d_inPos, d_outPos, d_predctTscan, d_predctFscan, d_predct, d_numPredctTelems, numElems);
    } 
    else {
      scatter<<<gridSize, blockSize>>>(d_outVals, d_inVals, d_predctTscan, d_predctFscan, d_predct, d_numPredctTelems, numElems);
      scatter<<<gridSize, blockSize>>>(d_outPos, d_inPos, d_predctTscan, d_predctFscan, d_predct, d_numPredctTelems, numElems);
    }
  }

  CUDA_CHECK(hipFree(d_predct));
  CUDA_CHECK(hipFree(d_predctTscan));
  CUDA_CHECK(hipFree(d_predctFscan));
  CUDA_CHECK(hipFree(d_numPredctTelems));
  CUDA_CHECK(hipFree(d_numPredctFelems));
  CUDA_CHECK(hipFree(d_blk_sums));
}

void printArray(int *a, int len, const char *fileName) {
    FILE *fptr;
    fptr = fopen(fileName, "w");
    for (int i = 0; i < len; i++) fprintf(fptr, "%d\n", a[i]);
    fclose(fptr);
}

int main(int argc, char** argv)  {
    
    const int DATASIZE = atoi(argv[1]); 
    const int numIterations = 1;

    float et = 0;
    float tmp_time = 0;
    
    hipEvent_t start, stop;
    hipEventCreate(&start); 
    hipEventCreate(&stop);
    
    int *h_data, *h_pos; // Host []s
    int *d_data, *d_pos, *d_data_out, *d_pos_out; // Dev []s
    unsigned int arrAlloc = DATASIZE * sizeof(int);
    
    h_data = (int*)malloc(arrAlloc);   // allocating memory for Hosts[]s
    h_pos = (int*)malloc(arrAlloc);

    hipMalloc((void**)&d_data, arrAlloc); // allocating memory for Dev[]s 
    hipMalloc((void**)&d_pos, arrAlloc); 
    hipMalloc((void**)&d_data_out, arrAlloc); 
    hipMalloc((void**)&d_pos_out, arrAlloc); 

    srand(time(NULL));
    for( int i = 0; i < DATASIZE; i++ ) {  // generating Host[] values
        h_data[i]  = rand() ; h_pos[i] = i;
    }
    printf("Sorting %d elements\n", DATASIZE); //DEBUG: printArray(h_data, DATASIZE, "input");
    
    CUDA_CHECK(hipMemcpy(d_pos, h_pos, arrAlloc, hipMemcpyHostToDevice));  // copy from Host to Dev
    hipDeviceSynchronize(); CUDA_CHECK(hipGetLastError());
    for (unsigned int i = 0; i < numIterations; i++) {

        hipEventRecord(start);    
        CUDA_CHECK(hipMemcpy(d_data, h_data, arrAlloc, hipMemcpyHostToDevice));  // copy from Host to Dev

        radixSort(d_data, d_pos, d_data_out, d_pos_out, DATASIZE);

        hipEventRecord(stop);    
        hipEventSynchronize(stop); 
        hipEventElapsedTime(&tmp_time, start, stop);
        hipDeviceSynchronize(); CUDA_CHECK(hipGetLastError());

        et += tmp_time;
    }

    CUDA_CHECK(hipMemcpy(h_data, d_data_out, arrAlloc, hipMemcpyDeviceToHost));  // copy from Dev to Host
    printf("Sorting %s\n", (std::is_sorted(h_data, h_data+DATASIZE) ? "succeed." : "FAILED.") );

/* DEBUG:
    CUDA_CHECK(hipMemcpy(h_pos, d_pos_out, arrAlloc, hipMemcpyDeviceToHost));  // copy from Dev to Host
    printArray(h_data, DATASIZE, "data_out"); printArray(h_pos, DATASIZE, "post_out");
*/
    tmp_time = et/1000/numIterations;
    printf("Throughput =%9.3lf MElements/s, Time = %.9lf ms\n",  1e-6*DATASIZE/tmp_time, tmp_time*1000);

    hipFree(d_data);
    hipFree(d_pos);
    hipDeviceReset();
    return 0;
}
