#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define DATASIZE  1000000
#define BLOCK_SIZE 128

void printArray(int a[],int n) {
    FILE *fptr;
    fptr = fopen("Brick_sort_result.txt", "w");

    for(int i=0; i < n; i++) {
        fprintf(fptr,"%d ",a[i]);
        
        if((i+1)%10 == 0)       
            fprintf(fptr,"\n");
    }
    fclose(fptr);
}

__global__ void oddevensort ( int * input, unsigned int size, int i )  {
    
    unsigned int myId = threadIdx.x + blockDim.x * blockIdx.x;

    int temp;
    int p;

    if(myId > size) return;

    if( i == 0 ) {
        p=myId*2;   // For even threads  
        // if(( myId % 2 == 0 && input[myId] > input[myId+1]))
        if(input[p]>input[p+1]) {
            temp = input[p+1];
            input[p+1] = input[p];
            input[p] = temp;
        }
    }
    else {
        p=myId*2+1; // for odd threads   
        // if(( myId % 2 != 0 && input[myId] > input[myId+1]))
        if(p<size-1){
            if(input[p]>input[p+1]) {
                temp = input[p+1];
                input[p+1] = input[p];
                input[p] = temp;
            }
        }
    }
    __syncthreads();
}

int main()  {
    
    float et=0;
    hipEvent_t start, stop;
    hipEventCreate(&start); 
    hipEventCreate(&stop);

    unsigned int arr_size = DATASIZE * sizeof(int);

    int h_input[DATASIZE], h_output[DATASIZE]; // Host []s
    int *d_input, *d_output; // Dev []s

    hipMalloc((void**)&d_input, arr_size ); // allocating memory for Dev[]s
    hipMalloc((void**)&d_output, arr_size );

    srand(time(NULL));

    for( int i = 0; i < DATASIZE; i++ ) h_input[i] = rand(); // generating Host [] values
    printf("Input array size : %d:\n",DATASIZE);
    
    if(DATASIZE<=100) {
        for( int i = 0; i < DATASIZE; i++ ) printf("%d ", h_input[i] );
    }
    printf("\n");

    int nthreads( BLOCK_SIZE );
    int nblocks( ceil((DATASIZE-1)/(float)BLOCK_SIZE) + 1 );

    hipEventRecord(start);     // start time

    hipMemcpy( d_input, h_input, arr_size, hipMemcpyHostToDevice); // copy from host to device

    for( int i=0; i<DATASIZE; i++) 
        oddevensort<<< nblocks,nthreads >>>( d_input, DATASIZE, i%2 );

    hipMemcpy( h_output, d_input, arr_size, hipMemcpyDeviceToHost);

    hipEventRecord(stop);     // end time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&et, start, stop);
    printf("Time is: %f\n",et);


    for(int i=0; i<DATASIZE-1; i++) { //  CHECK
        if(h_output[i]>h_output[i+1]) {
            printf("Sorting Failed!!");
            break;
        }
        else if(i == DATASIZE-2)
            printf("Sorting success!");
    }

    printArray(h_output,DATASIZE);
    printf("\n");

    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
